#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <pthread.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

#define PIXEL_SIZE 12
#define CHANNEL_SIZE 4
#define MAX_ITER 1000
#define NUM_THREADS 40



// define a struct used to hold complex numbers
typedef struct complex {
   double real;
   double imag;
} complex;


// define headers
void *thread_run();
void compute_pixel(int x, int y);
int compute_iterations(complex c);
complex f(complex z, complex c);
void set_color(int x, int y, int iterations);
complex pixel_to_complex(int pix_x, int pix_y);
void init_storage(void);
char *strcpy_no_nul(char *dest, const char *src);
void write_data_s(const int x, const int y, char *rgb);
void write_data_d(const int x, const int y, int r, int g, int b);
void save_file();
void display_status(double completed, double total, int length, int flush);


// initialize globals
double min_x, max_x, min_y, max_y;
int pix_width, pix_height;
void *storage;
//pthread_mutex_t mutex;
int current_row = 0;
void *GPUstorage;
int storageSize;


/* Main method */
int main(int argc, char **argv) {
	
	// need 7 arguments: program name and 6 parameters
	if (argc != 7) {
		printf("Invalid number of arguments (%d). Exiting main with status 1.\n", argc);
		exit(1);
	}

	// assign globals
    min_x = atof(argv[1]);
    min_y = atof(argv[2]);
    max_x = atof(argv[3]);
    max_y = atof(argv[4]);
    pix_width = atoi(argv[5]);
    pix_height = atoi(argv[6]);
	
	
	storageSize = PIXEL_SIZE*pix_width*pix_height;

	// initialize mutex
	//pthread_mutex_init(&mutex, NULL);

	// image data will be stored directly into memory
	// each pixel will have 3 channels (RGB) each 
	// requiring 4 ascii characters (000-255) and a space/newline
	// so we need 12 bytes/characters for each pixel plus 1 extra
	// byte to place a \0 null character to terminate the string
	// char is 1 byte in size
	// storage is a void* pointer to the memory block
	storage = malloc(storageSize + 1);
	hipMallocManaged(&GPUstorage, storageSize);

	// initialize the storage space to have necessary tabs, newlines,
	// spaces, and null character
	init_storage();
	
	// copying over to GPU
	hipMemcpy(GPUstorage, storage, storageSize, hipMemcpyHostToDevice);

	// initialize all threads
	/*pthread_t threads[NUM_THREADS];
	for (int t=0; t<NUM_THREADS; ++t) {
		pthread_create(&(threads[t]), NULL, thread_run, NULL);
	}*/
	
	// calling threads on thread_run function
	//thread_run<<<1, 1>>>(pix_height);

	// display status bar
	int status_length = 50;
	printf("\nComputing image...\n");
	while (current_row < pix_height) {
		display_status(current_row, pix_height, status_length, 0);
		usleep(10000);
	}
	display_status(current_row, pix_height, status_length, 1);

    // block on thread completion
	/*for (int t=0; t<NUM_THREADS; ++t) {
		pthread_join(threads[t], NULL);
	}*/
	
	// syncing threads
	hipDeviceSynchronize();
	
	// copy from GPU to CPU
	hipMemcpy(storage, GPUstorage, storageSize, hipMemcpyDeviceToHost);

	// save file
	printf("Saving image...\n");
	save_file();
	display_status(1, 1, status_length, 1);

	// free the memory that was allocated
	free(storage);
	hipFree(GPUstorage);
	printf("Finished.\n\n");
	
    return 0;
	
}


/* This method will be called by each thread that is created.
 * It will check the global current_row variable which is
 * mutex guarded and increment the variable and also run the
 * computation for the current row. It will do this as long
 * as there are rows that still need to be computed
 */
 __global__
void thread_run(int n, int pix_width) {
	while (1) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		int stride = blockDim.x * gridDim.x;
		
		for (int i = index; i < n; i += stride) {
			for (int x=0; x < pix_width; ++x) {
				compute_pixel(x, i);
			}
		}
		
		/*
		pthread_mutex_lock(&mutex);
			int row = current_row++;
		pthread_mutex_unlock(&mutex);
		if (row >= pix_height) break; 
		for (int x=0; x<pix_width; ++x) {
			compute_pixel(x, row);
		}
		*/
	}
	//pthread_exit(NULL);
}


/* This method will, given the x and y coordinates of a pixel,
 * convert the pixel to a complex value, compute the number
 * of iterations for escape, and write the corresponding color
 * into storage (in memory space). 
 */
 __global__
void compute_pixel(int x, int y) {
	complex c = pixel_to_complex(x,y);
	int iter = compute_iterations(c);
	set_color(x, y, iter);
}


/* Given a complex number, this function will return the
 * number of iterations needed for the value to escape from
 * the 2-unit circle centered at (0,0), unless that number
 * exceeds MAX_ITER in which case MAX_ITER will be returned
 */
__global__
int compute_iterations(complex c) {
	complex z = {0,0};
	
	int iter = 0;
	
	while (iter < MAX_ITER && z.real*z.real + z.imag*z.imag < 4) {
		z = f(z, c);
		++iter;
	}

	return iter;
}


/* This is the representation of the basic mandelbrot function */
__global__
complex f(complex z, complex c) {
	complex res;
	res.real = z.real*z.real - z.imag*z.imag + c.real;
	res.imag = 2*z.real*z.imag + c.imag;
	return res;
}


/* This function saves the corresponding color of a given pixel
 * to storage (memory space) based on its own internally defined
 * color mapping based on the number of iterations provided.
 */
 __global__
void set_color(int x, int y, int iterations) {
	int r, g, b;
	if (iterations == MAX_ITER) {
		r = 0;
		g = 0;
		b = 0;
	} else {
		int base = (int)(255*sqrt(1-pow((((double)iterations)-MAX_ITER)/MAX_ITER,2)));
		r = base;
		g = base;
		b = 0 + (255.0-0)/MAX_ITER*iterations;
	}
	
	write_data_d(x,y,r,g,b);
}


/* Given the x and y coordinates of a pixel, this function will
 * retun the complex value associated with the center of that
 * pixel. This value is based on the real and imaginary bounds
 * as well as the pixel dimensions provided via the terminal
 * arguments.
 */
 __global__
complex pixel_to_complex(int pix_x, int pix_y) {
    // TODO check this function for correctness

	double num_width = max_x - min_x;
	double num_height = max_y - min_y;
	complex c;

	c.real = min_x + num_width/pix_width*(pix_x+0.5);
	c.imag = max_y - num_height/pix_height*(pix_y+0.5);
	return c;
}


/*
 * This function will add spaces, newlines, and null
 * characters in the appropriate locations withint
 * the storage memory block.
 * 
 */
__global__
void init_storage(void) {
	int row_size = pix_width*PIXEL_SIZE;

	// change later to create spaces between 000_000_.....
	for (int loc=0; loc<=pix_width*pix_height*PIXEL_SIZE; ++loc) {
		*(char*)(storage + loc) = ' ';
	}
	
	for (int loc=1; loc<=pix_width*pix_height*PIXEL_SIZE/CHANNEL_SIZE; ++loc) {
		*(char*)(storage + loc*CHANNEL_SIZE - 1) = '\t';
	}

	for (int row=1; row<=pix_height; ++row) {
		*(char*)(storage+row*row_size - 1) = '\n';
	}

	*(char*)(storage + pix_width*pix_height*PIXEL_SIZE) = '\0';
}


/*
 * This will copy the string from src to dest, but
 * unlike the standard strcpy, the ending null character
 * will not be copied.
 */
 __global__
char *strcpy_no_nul(char *dest, const char *src)
{
  unsigned i;
  for (i=0; src[i] != '\0'; ++i)
    dest[i] = src[i];
  return dest;
}


/*
 * This will write the provided rgb color string to storage.
 * It is the callers responsibility to ensure that the string
 * is no more than 11 characters in length and that no rgb value
 * is less than 0 or greater than 255.
 * 
 * Example: write_data(1, 5, "100 008 89")
 */
 // We want this to be saved on GPU
 __global__
void write_data_s(const int x, const int y, char *rgb) {
	int row_size = pix_width*PIXEL_SIZE;
	//changing CPU storage to GPU storage
	void *addr = GPUstorage + x*PIXEL_SIZE + y*row_size;
	strcpy_no_nul(addr, rgb);
}


/*
 * This will write the provided rgb data to storage.
 * It is the callers responsibility to ensure that no rgb value
 * is less than 0 or greater than 255.
 * 
 * Example: write_data(1, 5, 100, 8, 89)
 */
 __global__
void write_data_d(const int x, const int y, int r, int g, int b) {
	char str[11];
	sprintf(str, "%d %d %d", r,g,b);
	write_data_s(x,y,str);
}


/*
 * This will save the storage to the file with the name provided.
 * The resulting file will be formatted as a netpbm (pbm) file.
 */
 __global__
void save_file() {
	
	FILE *file = fopen("~mandelbrot.pbm", "w");
	fprintf(file, "%s\n%d %d\n%d\n%s", 
	        "P3", pix_width, pix_height, 255, (char*)storage);
	fclose(file);

	// try to convert the temporary file to a jpg and if that succeeds
	// remove the temporary pbm file
	system("convert ~mandelbrot.pbm mandelbrot.jpg");
	if (access("mandelbrot.jpg", F_OK ) != -1 ) {
		// file exists
		system("rm ~mandelbrot.pbm");
	} else {
		// file does not exist likely because 'convert' is not installed
		printf("If you would like the output file converted from\
				a pbm file to a jpg file, install ImageMagick.");
	}
}


/* This function will display a status bar on the screen. It is the 
 * caller's responsibility to ensure that no other data is printed
 * to stdout, otherwise the results will not appear as desired.
 */
 __global__
void display_status(double completed, 
                    double total, 
                    int length, 
                    int flush) {
	if (flush) {
		completed = 1; 
		total = 1;
	}
	
	double progress = completed/total;
	int num_to_display = (int)(progress*length);
	char *bar = (char*)malloc(length+1);
	
	for (int i=0; i<length; ++i) {
		*(bar+i) = (i<num_to_display) ? '#' : '_';
	}
	bar[length] = '\0';
	
	printf("%s | %d%%%c",bar,(int)(progress*100),'\r');
	fflush(stdout);
	
	if (flush) printf("\n");
	
	free(bar);
}